
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<iostream>
#include <complex.h>
//#include<fftw3.h>
#include<hipfft/hipfftw.h>
#include<math.h>
#define ind_ele(n,i,j) n*(i)+j
#define get_start(l_s,k_s,x) (x-k_s>=0)?x-k_s:(l_s+(x-k_s))
#define inc(l_s,x) (x+1>=l_s)?0:x+1
#define NT 2
using namespace std;
void flip_col(double* mat,int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<n/2;j++){
            double temp;
            temp=mat[ind_ele(n,i,j)];
            mat[ind_ele(n,i,j)]=mat[ind_ele(n,i,n-1-j)];
            mat[ind_ele(n,i,n-1-j)]=temp;
        }
    }
}
void flip_row(double* mat,int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<n/2;j++){
            double temp;
            temp=mat[ind_ele(n,j,i)];
            mat[ind_ele(n,j,i)]=mat[ind_ele(n,n-j-1,i)];
            mat[ind_ele(n,n-j-1,i)]=temp;
        }
    }
}
double* create_matrix(int n){
    double* mat=new double[n*n];
    return mat;
}
void fill_random(double* mat,int n){
    for(int i=0;i<n*n;i++){
        mat[i]=rand()%10;
    }
}
void fill_constant(double* mat,double c,int n){
    for(int i=0;i<n*n;i++){
        //cout<<"inside iteration number "<<i<<endl;
        mat[i]=c;
    }
}
void fill_user(double* mat,int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            cout<<"--> ["<<i<<"]"<<"["<<j<<"]"<<endl;
            cin>>mat[ind_ele(n,i,j)];
        }
    }
}
void print_mat(double* arr,int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            cout<<arr[ind_ele(n,i,j)]<<" ";
        }
        cout<<endl;
    }
}
void print_comp_mat(fftwf_complex* arr,int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            cout<<"("<<arr[ind_ele(n,i,j)][0]<<","<<arr[ind_ele(n,i,j)][1]<<") ";
        }
        cout<<endl;
    
}
}
void write_to_real_mat(fftwf_complex* arr,double* arr_real,int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            arr_real[ind_ele(n,i,j)]=arr[ind_ele(n,i,j)][0];
        }
    }
}





void conv(double* layer,double* kernel,int layer_size,int kernel_size,double* output){
    flip_row(kernel,kernel_size);
    flip_col(kernel,kernel_size);
    for(int i=0;i<layer_size;i++){
        for(int j=0;j<layer_size;j++){
            int k_s=(kernel_size-1)/2;
            int l_row_start=get_start(layer_size,k_s,i);
            int l_col_start=get_start(layer_size,k_s,j);
            double sum=0;
            for(int ik=0;ik<kernel_size;ik++){
                l_col_start=get_start(layer_size,k_s,j);
                for(int jk=0;jk<kernel_size;jk++){
                    sum+=kernel[ind_ele(kernel_size,ik,jk)]*layer[ind_ele(layer_size,l_row_start,l_col_start)];
                    l_col_start=inc(layer_size,l_col_start);
                }
                l_row_start=inc(layer_size,l_row_start);
            }
            output[ind_ele(layer_size,i,j)]=sum;
        }
    }
}
double* preProcessKernel(double* plainKernel,int ini_size,int final_size){
    flip_col(plainKernel,ini_size);
    flip_row(plainKernel,ini_size);
    double* res_mat=create_matrix(final_size);
    fill_constant(res_mat,0,final_size);
    int ini_row=final_size-1;
    int ini_col=final_size-1;
    for(int i=0;i<ini_size;i++){
        ini_col=final_size-1;
        for(int j=0;j<ini_size;j++){
            res_mat[ind_ele(final_size,ini_row,ini_col)]=plainKernel[ind_ele(ini_size,i,j)];
            ini_col=inc(final_size,ini_col);
        } 
        ini_row=inc(final_size,ini_row);
    }
    return res_mat;
}
double* preProcessKernel_static(double* plainKernel,double* res_mat,int ini_size,int final_size){
    flip_col(plainKernel,ini_size);
    flip_row(plainKernel,ini_size);
    fill_constant(res_mat,0,final_size);
    int ini_row=final_size-1;
    int ini_col=final_size-1;
    for(int i=0;i<ini_size;i++){
        ini_col=final_size-1;
        for(int j=0;j<ini_size;j++){
            res_mat[ind_ele(final_size,ini_row,ini_col)]=plainKernel[ind_ele(ini_size,i,j)];
            ini_col=inc(final_size,ini_col);
        }
        ini_row=inc(final_size,ini_row);
    }
    return res_mat;
}

void mul(fftwf_complex* A,fftwf_complex* B,fftwf_complex* output,int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            output[ind_ele(n,i,j)][0]=(((A[ind_ele(n,i,j)][0])*(B[ind_ele(n,i,j)][0]))-((A[ind_ele(n,i,j)][1])*(B[ind_ele(n,i,j)][1])));  
            output[ind_ele(n,i,j)][1]=(((A[ind_ele(n,i,j)][0])*(B[ind_ele(n,i,j)][1]))+((A[ind_ele(n,i,j)][1])*(B[ind_ele(n,i,j)][0])));
        }
    }
}
void scale_comp(fftwf_complex* mat,int n){
    int nn=n*n;
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            mat[ind_ele(n,i,j)][0]=mat[ind_ele(n,i,j)][0]/nn;
        }
    }
}

fftwf_complex* get_complex_representation(double* mat,int n){
    fftwf_complex* res=new fftwf_complex[n*n];
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            res[ind_ele(n,i,j)][0]=mat[ind_ele(n,i,j)];
            res[ind_ele(n,i,j)][1]=0;
        }
    }
    return res;
}
//------------------------------ al kernel methods ---------------------------------------------------
__global__ void to_real_p(fftwf_complex* arr,double* arr_real,int n){
    int tid=threadIdx.x;
    int row_per_thread=n/NT;
    int start_row=tid*row_per_thread;//start row is included.
    int end_row=start_row+row_per_thread;//end row is not included.
    if(tid==NT-1){//last threads flow
        end_row=n;
    }
    for(int i=start_row;i<end_row;i++){
        for(int j=0;j<n;j++){
            arr_real[ind_ele(n,i,j)]=arr[ind_ele(n,i,j)][0];
        }
    }
}
__global__ void scale_p(fftwf_complex* mat,int n){
    int nn=n*n;
    int tid=threadIdx.x;
    int row_per_thread=n/NT;
    int start_row=tid*row_per_thread;//start row is included.
    int end_row=start_row+row_per_thread;//end row is not included.
    if(tid==NT-1){//last threads flow
        end_row=n;
    }
    for(int i=start_row;i<end_row;i++){
        for(int j=0;j<n;j++){
            mat[ind_ele(n,i,j)][0]=mat[ind_ele(n,i,j)][0]/nn;
        }
    }
}
__global__ void conv_complex(double* src,fftwf_complex* des,int n){
    //NT
    //printf("starting the complex converstion\n");
    int tid=threadIdx.x;
    //printf("tid is %d\n",tid);
    //printf("the value of n is %d and value fo NT is",n);
    if(NT==2){
        printf("NT is 2");
    }
    int row_per_thread=n/NT;
    int start_row=tid*row_per_thread;//start row is included.
    int end_row=start_row+row_per_thread;//end row is not included.
    //printf("start row is %d\n",start_row);
    //printf("end row is %d\n",end_row);
    if(tid==NT-1){//last threads flow
        end_row=n;
    }
    for(int i=start_row;i<end_row;i++){
        for(int j=0;j<n;j++){
            //printf("%d %d %d\n",tid,i,j);
            des[ind_ele(n,i,j)][0]=src[ind_ele(n,i,j)];
        }
    }
}

__global__ void poin_mul_parallel(fftwf_complex* A,fftwf_complex* B,fftwf_complex* output,int n){
    int tid=threadIdx.x;
    int row_per_thread=n/NT;
    int start_row=tid*row_per_thread;//start row is included.
    int end_row=start_row+row_per_thread;//end row is not included.
    if(tid==NT-1){//last threads flow
        end_row=n;
    }
    for(int i=start_row;i<end_row;i++){
        for(int j=0;j<n;j++){
            //c[ind_ele(n,i,j)]=a[ind_ele(n,i,j)]*b[ind_ele(n,i,j)];
            output[ind_ele(n,i,j)][0]=(((A[ind_ele(n,i,j)][0])*(B[ind_ele(n,i,j)][0]))-((A[ind_ele(n,i,j)][1])*(B[ind_ele(n,i,j)][1])));  
            output[ind_ele(n,i,j)][1]=(((A[ind_ele(n,i,j)][0])*(B[ind_ele(n,i,j)][1]))+((A[ind_ele(n,i,j)][1])*(B[ind_ele(n,i,j)][0])));
        }
    }
}
__global__ void sumP(double* arr,double* ac_sum,int n){
    double sum=0;
    int tid=threadIdx.x;
    int row_per_thread=n/NT;
    __shared__ double cache[NT];
    int start_row=tid*row_per_thread;//start row is included.
    int end_row=start_row+row_per_thread;//end row is not included.
    if(tid==NT-1){//last threads flow
        end_row=n;
    }
    for(int i=start_row;i<end_row;i++){
        for(int j=0;j<n;j++){
            sum+=arr[ind_ele(n,i,j)];
        }
    }
    cache[tid]=sum;
    __syncthreads();
    *ac_sum=0;
    if(tid==0){
        for(int i=0;i<NT;i++){
            *ac_sum+=cache[i];
        }
    }
}
struct convolution_enviorment{
    //things required for forward propogation
    double* layer;//d
    double* kernel;//d
    double* d_layer;//d
    double* d_kernel;//d
    fftwf_complex* d_comp_layer;//d
    fftwf_complex* d_comp_kernel;//d
    fftwf_complex* d_fft_layer;//d
    fftwf_complex* d_fft_kernel;//d
    fftwf_complex* d_fft_mul;//d
    fftwf_complex* d_mul;//d
    double* d_real_mul;//d
    double* d_predicted;//d 
    double* predicted;//d

    fftwf_complex* fft_layer;
    fftwf_complex* fft_kernel;
    fftwf_complex* fft_mul;
    fftwf_complex* mul;
    fftwf_complex* fft_gradient;
    fftwf_complex* fft_kernel_gradient;
    fftwf_complex* kernel_gradient;
    double* real_mul;
    int filter_size;
    fftwf_plan get_layer;
    fftwf_plan get_kernel;
    fftwf_plan getBack_mul;
    fftwf_plan get_gradient;
    fftwf_plan getBackkernel_gradient;
    double* kernel_gradient_real;
    double* gradients;
    double learning_rate;
};
convolution_enviorment create_convolution_enviormentParellel(int fs){
    convolution_enviorment ce;
    ce.filter_size=fs;
    ce.layer=new double[fs*fs];
    ce.kernel=new double[fs*fs];
    hipMalloc((void**)&ce.d_layer,sizeof(double)*fs*fs);             //things may get pointerise.
    hipMalloc((void**)&ce.d_kernel,sizeof(double)*fs*fs);
    hipMalloc((void**)&ce.d_fft_layer,sizeof(fftwf_complex)*fs*fs);
    hipMalloc((void**)&ce.d_fft_kernel,sizeof(fftwf_complex)*fs*fs);
    hipMalloc((void**)&ce.d_fft_mul,sizeof(fftwf_complex)*fs*fs);
    hipMalloc((void**)&ce.d_mul,sizeof(fftwf_complex)*fs*fs);
    hipMalloc((void**)&ce.d_comp_kernel,sizeof(fftwf_complex)*fs*fs);
    hipMalloc((void**)&ce.d_comp_layer,sizeof(fftwf_complex)*fs*fs);
    hipMalloc((void**)&ce.d_real_mul,sizeof(double)*fs*fs);
    hipMalloc((void**)&ce.d_predicted,sizeof(double));
    ce.predicted=new double();
    *ce.predicted=10;
    printf("while creation pridected value is %f\n",*ce.predicted);
    return ce;
}
void forwardPass_parallel(convolution_enviorment ce){
    cout<<"starting the routine"<<endl;
    int fs=ce.filter_size*ce.filter_size;
    hipMemcpy(ce.d_layer,ce.layer,fs,hipMemcpyHostToDevice);
    hipMemcpy(ce.d_kernel,ce.kernel,fs,hipMemcpyHostToDevice);
    cout<<"senf the kernel and layer to the device"<<endl;
    conv_complex<<<1,NT>>>(ce.d_layer,ce.d_comp_layer,ce.filter_size);
    conv_complex<<<1,NT>>>(ce.d_kernel,ce.d_comp_kernel,ce.filter_size);
    cout<<"convertin them to complex"<<endl;
    ce.get_layer=fftwf_plan_dft_2d(ce.filter_size,ce.filter_size,ce.d_comp_layer,ce.d_fft_layer,FFTW_FORWARD,FFTW_ESTIMATE);
    ce.get_kernel=fftwf_plan_dft_2d(ce.filter_size,ce.filter_size,ce.d_comp_kernel,ce.d_fft_kernel,FFTW_FORWARD,FFTW_ESTIMATE);
    fftwf_execute(ce.get_layer);
    fftwf_execute(ce.get_kernel);    
    hipDeviceSynchronize();
    cout<<"doing the fft transform"<<endl;
    poin_mul_parallel<<<1,NT>>>(ce.d_fft_kernel,ce.d_fft_layer,ce.d_fft_mul,ce.filter_size);
    hipDeviceSynchronize();
    cout<<"done point wise multiplication"<<endl;
    ce.getBack_mul=fftwf_plan_dft_2d(ce.filter_size,ce.filter_size,ce.d_fft_mul,ce.d_mul,FFTW_BACKWARD,FFTW_ESTIMATE);
    fftwf_execute(ce.getBack_mul);
    hipDeviceSynchronize();
    cout<<"inverse fft done"<<endl;
    scale_p<<<1,NT>>>(ce.d_mul,ce.filter_size);
    
    hipDeviceSynchronize();
    cout<<"scalling is done"<<endl;
    to_real_p<<<1,NT>>>(ce.d_mul,ce.d_real_mul,ce.filter_size);
    
    hipDeviceSynchronize();
    cout<<"convertion to real done"<<endl;
    sumP<<<1,NT>>>(ce.d_real_mul,ce.d_predicted,ce.filter_size);
    hipMemcpy(ce.predicted,ce.d_predicted,sizeof(double),hipMemcpyDeviceToHost);
    cout<<"summation doen and copy to host done"<<endl;
}

void insialize_backProp(convolution_enviorment* ce,double lr){
    int tt=ce->filter_size;
    tt=tt*tt;
    ce->fft_gradient = new fftwf_complex[tt];
    ce->fft_kernel_gradient=new fftwf_complex[tt];
    ce->kernel_gradient=new fftwf_complex[tt];
    //ce->aux_mat=new double[tt];
    //ce->aux_mat2=new double[tt];
    ce->kernel_gradient_real=new double[tt];
    ce->gradients=new double[tt];
    ce->learning_rate=lr;
}
convolution_enviorment create_convolution_enviorment(int fs){
    convolution_enviorment myenv;
    myenv.filter_size=fs;
    myenv.layer=create_matrix(fs);
    myenv.kernel=create_matrix(fs);
    myenv.real_mul=create_matrix(fs);
    myenv.fft_kernel=new fftwf_complex[fs*fs];
    myenv.fft_layer=new fftwf_complex[fs*fs];
    myenv.fft_mul=new fftwf_complex[fs*fs];
    myenv.mul=new fftwf_complex[fs*fs];
    //myenv.aux_mat=create_matrix(myenv.filter_size);
    return myenv;
}
void mat_mul(double* a,double* b,double* res,int n){
    for (int i = 0; i<n;i++) {
        for (int j =0; j<n;j++) {
            for (int k=0;k<n;k++) {
                res[ind_ele(n,i,j)]+=a[ind_ele(n,i,k)]*b[ind_ele(n,k,j)];
            }
        }
    }
}
/*
void mat_one_minus_this(double* a,double* res,int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            res[ind_ele(n,i,j)]=1-a[ind_ele(n,i,j)];
        }
    }
}*/

void updatekernel(convolution_enviorment ce){
    int n=ce.filter_size;
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            ce.kernel[ind_ele(n,i,j)]-=(ce.learning_rate*ce.kernel_gradient_real[ind_ele(n,i,j)]);
        }
    }
}

/*
void relu_derivative(double* in,double* out,int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            if(in[ind_ele(n,i,j)]>0){
                out[ind_ele(n,i,j)]=1;
            }else{
                out[ind_ele(n,i,j)]=0;
            }
        }
    }
}*/

//code to update the kernel is not included in the backprop.

void backPropogate(convolution_enviorment ce,double actual_value){
    /*
    double temp=2*(ce.predicted-actual_value);
    fill_constant(ce.gradients,temp,ce.filter_size);
    ce.get_gradient=fftwf_plan_dft_2d(ce.filter_size,ce.filter_size,get_complex_representation(ce.gradients,ce.filter_size),ce.fft_gradient,FFTW_FORWARD,FFTW_ESTIMATE);
    fftwf_execute(ce.get_gradient);
    mul(ce.fft_gradient,ce.fft_layer,ce.fft_kernel_gradient,ce.filter_size);
    ce.getBackkernel_gradient=fftwf_plan_dft_2d(ce.filter_size,ce.filter_size,ce.fft_kernel_gradient,ce.kernel_gradient,FFTW_BACKWARD,FFTW_ESTIMATE);
    fftwf_execute(ce.getBackkernel_gradient);
    scale_comp(ce.kernel_gradient,ce.filter_size);
    write_to_real_mat(ce.kernel_gradient,ce.kernel_gradient_real,ce.filter_size);*/
}


/*
//activation functioin needs to added here.
void forwardPass_convolution_enviorment(convolution_enviorment ce){
    ce.get_layer=fftwf_plan_dft_2d(ce.filter_size,ce.filter_size,get_complex_representation(ce.layer,ce.filter_size),ce.fft_layer,FFTW_FORWARD,FFTW_ESTIMATE);
    ce.get_kernel=fftwf_plan_dft_2d(ce.filter_size,ce.filter_size,get_complex_representation(ce.kernel,ce.filter_size),ce.fft_kernel,FFTW_FORWARD,FFTW_ESTIMATE);
    
    fftwf_execute(ce.get_layer);
    
    fftwf_execute(ce.get_kernel);
    
    mul(ce.fft_layer,ce.fft_kernel,ce.fft_mul,ce.filter_size);
    
    ce.getBack_mul=fftwf_plan_dft_2d(ce.filter_size,ce.filter_size,ce.fft_mul,ce.mul,FFTW_BACKWARD,FFTW_ESTIMATE);
    
    fftwf_execute(ce.getBack_mul);
    
    scale_comp(ce.mul,ce.filter_size);
    
    write_to_real_mat(ce.mul,ce.real_mul,ce.filter_size);
}
*/
//this method is incomplete.
void destroy_convolutional_enviorment(convolution_enviorment ce){
    delete[] ce.fft_kernel;
    delete[] ce.fft_layer;
    delete[] ce.fft_mul;
    delete[] ce.mul;
    delete[] ce.layer;
    delete[] ce.kernel;
    delete[] ce.real_mul;
    //and destroy other plans.
} 


//method will get changes a lot
double psudoFullyConnected(double* output,int n){
    double sum=0;
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            sum+=output[ind_ele(n,i,j)];
        }
    }
    return sum;
}
void messaure_normal_time(){
    int layer_length=512;
    double* layer=create_matrix(layer_length);
    fill_random(layer,layer_length);
    double* output=create_matrix(layer_length);
    for(int kernel_length=3;kernel_length<512;kernel_length+=2){
        double* kernel=create_matrix(kernel_length);
        fill_random(kernel,kernel_length);
        clock_t start_time;
        clock_t end_time;
        start_time=clock();
        for(int i=0;i<10;i++){
            conv(layer,kernel,layer_length,kernel_length,output);
        }
        end_time=clock();
        double clock_taken=double(end_time - start_time);
        clock_taken=clock_taken/10;
        double time_taken=clock_taken/double(CLOCKS_PER_SEC);
        cout<<kernel_length<<" "<<time_taken<<endl;
        delete kernel;
    }
}

//before we need to set the kernel we need to preprocess teh kernel
//we need to set the layer
//data set is not coded in it yet
/*
void trainIteration(convolution_enviorment ce){
    cout<<"doing forward pass"<<endl;
    forwardPass_convolution_enviorment(ce);
    //cout<<"output after forward propogation is "<<endl;
    //print_mat(ce.real_mul,ce.filter_size);
    cout<<"applying fully connected layer"<<endl;
    ce.predicted=psudoFullyConnected(ce.real_mul,ce.filter_size);
    cout<<"the predicted value is "<<ce.predicted<<endl;
    cout<<"the error is "<<(ce.predicted-20)*(ce.predicted-20)<<endl;
    backPropogate(ce,20);
    cout<<"back propogation done"<<endl;
    updatekernel(ce);
    cout<<"kernel updated"<<endl;
}*/
int main(){
    cout<<"starting the main program"<<endl;
    convolution_enviorment ce=create_convolution_enviormentParellel(10);
    double* k=create_matrix(5);
    fill_random(ce.layer,10);
    //fill_random(k,5);
    fill_constant(k,20,5);
    preProcessKernel_static(k,ce.kernel,5,10);
    cout<<"till forwar pass everything is cool"<<endl;
    forwardPass_parallel(ce);
    cout<<"the pridicted value is "<<*ce.predicted<<endl;
    cout<<"working till ending"<<endl;
    return 0;
}






//layer=[[1,1,1,2,1],[8,3,2,6,2],[4,5,6,3,3],[1,1,1,1,1],[3,3,3,4,5]]
//kernel=[[1,1,2],[3,1,2],[4,5,5]]
/*

convolution_enviorment ce=create_convolution_enviorment(5);
    double* kernel=create_matrix(3);
    cout<<"enter the value in the layer"<<endl;
    fill_user(ce.layer,5);
    cout<<"enter the value in the kernel"<<endl;
    fill_user(kernel,3);
    preProcessKernel_static(kernel,ce.kernel,3,5);
    cout<<"doing the forward pass"<<endl;
    forwardPass_convolution_enviorment(ce);
    cout<<"the output generated by the forward pass is "<<endl;
    print_mat(ce.real_mul,ce.filter_size);
    print_comp_mat(ce.mul,ce.filter_size);
    return 0;









double* l=create_matrix(5);
    double* k=create_matrix(3);
    cout<<"enter the elements of the layler"<<endl;
    fill_user(l,5);
    cout<<"enter the elemnts of the kernel"<<endl;
    fill_user(k,3);
    double* ck=preProcessKernel(k,3,5);
    fftwf_complex* output=new fftwf_complex[5*5];
    double* output2=create_matrix(5);
    fft_conv(l,ck,5,output);
    conv(l,k,5,3,output2);
    cout<<"the convolution operation gives"<<endl;
    print_comp_mat(output,5);
    print_mat(output2,5);

*/