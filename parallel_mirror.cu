
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<iostream>
#include <complex.h>
//#include<fftw3.h>
#include<hipfft/hipfftw.h>
#include<math.h>
#define ind_ele(n,i,j) n*(i)+j
#define get_start(l_s,k_s,x) (x-k_s>=0)?x-k_s:(l_s+(x-k_s))
#define inc(l_s,x) (x+1>=l_s)?0:x+1
#define NT 2
using namespace std;
void flip_col(double* mat,int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<n/2;j++){
            double temp;
            temp=mat[ind_ele(n,i,j)];
            mat[ind_ele(n,i,j)]=mat[ind_ele(n,i,n-1-j)];
            mat[ind_ele(n,i,n-1-j)]=temp;
        }
    }
}
void flip_row(double* mat,int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<n/2;j++){
            double temp;
            temp=mat[ind_ele(n,j,i)];
            mat[ind_ele(n,j,i)]=mat[ind_ele(n,n-j-1,i)];
            mat[ind_ele(n,n-j-1,i)]=temp;
        }
    }
}
double* create_matrix(int n){
    double* mat=new double[n*n];
    return mat;
}
void fill_random(double* mat,int n){
    for(int i=0;i<n*n;i++){
        mat[i]=rand()%10;
    }
}
void fill_constant(double* mat,double c,int n){
    for(int i=0;i<n*n;i++){
        //cout<<"inside iteration number "<<i<<endl;
        mat[i]=c;
    }
}
void fill_user(double* mat,int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            cout<<"--> ["<<i<<"]"<<"["<<j<<"]"<<endl;
            cin>>mat[ind_ele(n,i,j)];
        }
    }
}
void print_mat(double* arr,int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            cout<<arr[ind_ele(n,i,j)]<<" ";
        }
        cout<<endl;
    }
}
void print_comp_mat(fftwf_complex* arr,int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            cout<<"("<<arr[ind_ele(n,i,j)][0]<<","<<arr[ind_ele(n,i,j)][1]<<") ";
        }
        cout<<endl;
    
}
}
void write_to_real_mat(fftwf_complex* arr,double* arr_real,int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            arr_real[ind_ele(n,i,j)]=arr[ind_ele(n,i,j)][0];
        }
    }
}





void conv(double* layer,double* kernel,int layer_size,int kernel_size,double* output){
    flip_row(kernel,kernel_size);
    flip_col(kernel,kernel_size);
    for(int i=0;i<layer_size;i++){
        for(int j=0;j<layer_size;j++){
            int k_s=(kernel_size-1)/2;
            int l_row_start=get_start(layer_size,k_s,i);
            int l_col_start=get_start(layer_size,k_s,j);
            double sum=0;
            for(int ik=0;ik<kernel_size;ik++){
                l_col_start=get_start(layer_size,k_s,j);
                for(int jk=0;jk<kernel_size;jk++){
                    sum+=kernel[ind_ele(kernel_size,ik,jk)]*layer[ind_ele(layer_size,l_row_start,l_col_start)];
                    l_col_start=inc(layer_size,l_col_start);
                }
                l_row_start=inc(layer_size,l_row_start);
            }
            output[ind_ele(layer_size,i,j)]=sum;
        }
    }
}
double* preProcessKernel(double* plainKernel,int ini_size,int final_size){
    flip_col(plainKernel,ini_size);
    flip_row(plainKernel,ini_size);
    double* res_mat=create_matrix(final_size);
    fill_constant(res_mat,0,final_size);
    int ini_row=final_size-1;
    int ini_col=final_size-1;
    for(int i=0;i<ini_size;i++){
        ini_col=final_size-1;
        for(int j=0;j<ini_size;j++){
            res_mat[ind_ele(final_size,ini_row,ini_col)]=plainKernel[ind_ele(ini_size,i,j)];
            ini_col=inc(final_size,ini_col);
        } 
        ini_row=inc(final_size,ini_row);
    }
    return res_mat;
}
double* preProcessKernel_static(double* plainKernel,double* res_mat,int ini_size,int final_size){
    flip_col(plainKernel,ini_size);
    flip_row(plainKernel,ini_size);
    fill_constant(res_mat,0,final_size);
    int ini_row=final_size-1;
    int ini_col=final_size-1;
    for(int i=0;i<ini_size;i++){
        ini_col=final_size-1;
        for(int j=0;j<ini_size;j++){
            res_mat[ind_ele(final_size,ini_row,ini_col)]=plainKernel[ind_ele(ini_size,i,j)];
            ini_col=inc(final_size,ini_col);
        }
        ini_row=inc(final_size,ini_row);
    }
    return res_mat;
}

void mul(fftwf_complex* A,fftwf_complex* B,fftwf_complex* output,int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            output[ind_ele(n,i,j)][0]=(((A[ind_ele(n,i,j)][0])*(B[ind_ele(n,i,j)][0]))-((A[ind_ele(n,i,j)][1])*(B[ind_ele(n,i,j)][1])));  
            output[ind_ele(n,i,j)][1]=(((A[ind_ele(n,i,j)][0])*(B[ind_ele(n,i,j)][1]))+((A[ind_ele(n,i,j)][1])*(B[ind_ele(n,i,j)][0])));
        }
    }
}
void scale_comp(fftwf_complex* mat,int n){
    int nn=n*n;
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            mat[ind_ele(n,i,j)][0]=mat[ind_ele(n,i,j)][0]/nn;
        }
    }
}

fftwf_complex* get_complex_representation(double* mat,int n){
    fftwf_complex* res=new fftwf_complex[n*n];
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            res[ind_ele(n,i,j)][0]=mat[ind_ele(n,i,j)];
            res[ind_ele(n,i,j)][1]=0;
        }
    }
    return res;
}
//------------------------------ al kernel methods ---------------------------------------------------
__global__ void to_real_p(fftwf_complex* arr,double* arr_real,int n){
    int tid=threadIdx.x;
    int row_per_thread=n/NT;
    int start_row=tid*row_per_thread;//start row is included.
    int end_row=start_row+row_per_thread;//end row is not included.
    if(tid==NT-1){//last threads flow
        end_row=n;
    }
    for(int i=start_row;i<end_row;i++){
        for(int j=0;j<n;j++){
            arr_real[ind_ele(n,i,j)]=arr[ind_ele(n,i,j)][0];
        }
    }
}
__global__ void scale_p(fftwf_complex* mat,int n){
    int nn=n*n;
    int tid=threadIdx.x;
    int row_per_thread=n/NT;
    int start_row=tid*row_per_thread;//start row is included.
    int end_row=start_row+row_per_thread;//end row is not included.
    if(tid==NT-1){//last threads flow
        end_row=n;
    }
    for(int i=start_row;i<end_row;i++){
        for(int j=0;j<n;j++){
            mat[ind_ele(n,i,j)][0]=mat[ind_ele(n,i,j)][0]/nn;
        }
    }
}
__global__ void conv_complex(double* src,fftwf_complex* des,int n){
    //NT
    //printf("starting the complex converstion\n");
    int tid=threadIdx.x;
    int row_per_thread=n/NT;
    int start_row=tid*row_per_thread;//start row is included.
    int end_row=start_row+row_per_thread;//end row is not included.
    //printf("start row is %d\n",start_row);
    //printf("end row is %d\n",end_row);
    if(tid==NT-1){//last threads flow
        end_row=n;
    }
    for(int i=start_row;i<end_row;i++){
        for(int j=0;j<n;j++){
            //printf("%d %d %d\n",tid,i,j);
            des[ind_ele(n,i,j)][0]=src[ind_ele(n,i,j)];
        }
    }
}

__global__ void poin_mul_parallel(fftwf_complex* A,fftwf_complex* B,fftwf_complex* output,int n){
    int tid=threadIdx.x;
    int row_per_thread=n/NT;
    int start_row=tid*row_per_thread;//start row is included.
    int end_row=start_row+row_per_thread;//end row is not included.
    if(tid==NT-1){//last threads flow
        end_row=n;
    }
    for(int i=start_row;i<end_row;i++){
        for(int j=0;j<n;j++){
            //c[ind_ele(n,i,j)]=a[ind_ele(n,i,j)]*b[ind_ele(n,i,j)];
            output[ind_ele(n,i,j)][0]=(((A[ind_ele(n,i,j)][0])*(B[ind_ele(n,i,j)][0]))-((A[ind_ele(n,i,j)][1])*(B[ind_ele(n,i,j)][1])));  
            output[ind_ele(n,i,j)][1]=(((A[ind_ele(n,i,j)][0])*(B[ind_ele(n,i,j)][1]))+((A[ind_ele(n,i,j)][1])*(B[ind_ele(n,i,j)][0])));
        }
    }
}
__global__ void sumP(double* arr,double* ac_sum,int n){
    double sum=0;
    int tid=threadIdx.x;
    int row_per_thread=n/NT;
    __shared__ double cache[NT];
    int start_row=tid*row_per_thread;//start row is included.
    int end_row=start_row+row_per_thread;//end row is not included.
    if(tid==NT-1){//last threads flow
        end_row=n;
    }
    for(int i=start_row;i<end_row;i++){
        for(int j=0;j<n;j++){
            sum+=arr[ind_ele(n,i,j)];
        }
    }
    cache[tid]=sum;
    __syncthreads();
    *ac_sum=0;
    if(tid==0){
        for(int i=0;i<NT;i++){
            *ac_sum+=cache[i];
        }
    }
}
__global__ void fill_constant_parallel(double* grad,double temp,int n){
    int tid=threadIdx.x;
    int row_per_thread=n/NT;
    int start_row=tid*row_per_thread;//start row is included.
    int end_row=start_row+row_per_thread;//end row is not included.
    if(tid==NT-1){//last threads flow
        end_row=n;
    }
    for(int i=start_row;i<end_row;i++){
        for(int j=0;j<n;j++){
            grad[ind_ele(n,i,j)]=temp;
        }
    }
}
__global__ void updatekernel(double* kernel,double* kernel_grad,double lr,int n){
    int tid=threadIdx.x;
    int row_per_thread=n/NT;
    int start_row=tid*row_per_thread;//start row is included.
    int end_row=start_row+row_per_thread;//end row is not included.
    if(tid==NT-1){//last threads flow
        end_row=n;
    }
    for(int i=start_row;i<end_row;i++){
        for(int j=0;j<n;j++){
            kernel[ind_ele(n,i,j)]=kernel[ind_ele(n,i,j)]-(lr*kernel_grad[ind_ele(n,i,j)]);
        }
    }
}
struct convolution_enviorment{
    //things required for forward propogation
    double* layer;//d
    double* kernel;//d
    double* d_layer;//d
    double* d_kernel;//d
    fftwf_complex* d_comp_layer;//d
    fftwf_complex* d_comp_kernel;//d
    fftwf_complex* d_fft_layer;//d
    fftwf_complex* d_fft_kernel;//d
    fftwf_complex* d_fft_mul;//d
    fftwf_complex* d_mul;//d
    double* d_real_mul;//d
    double* d_predicted;//d 
    double* predicted;//d
    int filter_size;
    fftwf_plan get_layer;
    fftwf_plan get_kernel;
    fftwf_plan getBack_mul;

    double* d_grad;//d
    fftwf_complex* d_comp_grad;//d
    fftwf_complex* d_comp_fft_grad;//dd
    fftwf_complex* d_comp_fft_kernel_grad;//d
    fftwf_complex* d_comp_kernel_grad;//d
    double* d_kernel_grad;
    fftwf_plan get_gradient;
    fftwf_plan getBackkernel_gradient;
    double learning_rate;
};
/*
void insialize_backProp(convolution_enviorment* ce,double lr){
    int tt=ce->filter_size;
    tt=tt*tt;
    ce->fft_gradient = new fftwf_complex[tt];
    ce->fft_kernel_gradient=new fftwf_complex[tt];
    ce->kernel_gradient=new fftwf_complex[tt];
    //ce->aux_mat=new double[tt];
    //ce->aux_mat2=new double[tt];
    ce->kernel_gradient_real=new double[tt];
    ce->gradients=new double[tt];
    ce->learning_rate=lr;
}*/
convolution_enviorment create_convolution_enviormentParellel(int fs){
    convolution_enviorment ce;
    ce.filter_size=fs;
    ce.layer=new double[fs*fs];
    ce.kernel=new double[fs*fs];
    hipMalloc((void**)&ce.d_layer,sizeof(double)*fs*fs);             //things may get pointerise.
    hipMalloc((void**)&ce.d_kernel,sizeof(double)*fs*fs);
    hipMalloc((void**)&ce.d_fft_layer,sizeof(fftwf_complex)*fs*fs);
    hipMalloc((void**)&ce.d_fft_kernel,sizeof(fftwf_complex)*fs*fs);
    hipMalloc((void**)&ce.d_fft_mul,sizeof(fftwf_complex)*fs*fs);
    hipMalloc((void**)&ce.d_mul,sizeof(fftwf_complex)*fs*fs);
    hipMalloc((void**)&ce.d_comp_kernel,sizeof(fftwf_complex)*fs*fs);
    hipMalloc((void**)&ce.d_comp_layer,sizeof(fftwf_complex)*fs*fs);
    hipMalloc((void**)&ce.d_real_mul,sizeof(double)*fs*fs);
    hipMalloc((void**)&ce.d_predicted,sizeof(double));
    ce.predicted=new double();
    *ce.predicted=10;
    return ce;
}
void ini_bp(convolution_enviorment* ce,double le){
    int tt=ce->filter_size;
    tt=tt*tt;
    hipMalloc((void**)&(ce->d_grad),sizeof(double)*tt);
    hipMalloc((void**)&(ce->d_kernel_grad),sizeof(double)*tt);
    hipMalloc((void**)&(ce->d_comp_grad),sizeof(fftwf_complex)*tt);
    hipMalloc((void**)&(ce->d_comp_fft_grad),sizeof(fftwf_complex)*tt);
    hipMalloc((void**)&(ce->d_comp_fft_kernel_grad),sizeof(fftwf_complex)*tt);
    hipMalloc((void**)&(ce->d_comp_kernel_grad),sizeof(fftwf_complex)*tt);
    ce->learning_rate=le;
}
void backPropogate(convolution_enviorment ce,double actual_value){
    //cout<<"mse error is "<<(*(ce.predicted)-actual_value)*(*(ce.predicted)-actual_value)<<endl;
    //cout<<"entering backprog"<<endl;
    double temp=2*(*(ce.predicted)-actual_value);
    fill_constant_parallel<<<1,NT>>>(ce.d_grad,temp,ce.filter_size);
    hipDeviceSynchronize();
    //cout<<"filling of constant done"<<endl;
    conv_complex<<<1,NT>>>(ce.d_grad,ce.d_comp_grad,ce.filter_size);
    hipDeviceSynchronize();
    //cout<<"convertion into complex"<<endl;
    ce.get_gradient=fftwf_plan_dft_2d(ce.filter_size,ce.filter_size,ce.d_comp_grad,ce.d_comp_fft_grad,FFTW_FORWARD,FFTW_ESTIMATE);
    fftwf_execute(ce.get_gradient);
    poin_mul_parallel<<<1,NT>>>(ce.d_comp_fft_grad,ce.d_fft_layer,ce.d_comp_fft_kernel_grad,ce.filter_size);
    hipDeviceSynchronize();
    ce.getBackkernel_gradient=fftwf_plan_dft_2d(ce.filter_size,ce.filter_size,ce.d_comp_fft_kernel_grad,ce.d_comp_kernel_grad,FFTW_BACKWARD,FFTW_ESTIMATE);
    fftwf_execute(ce.getBackkernel_gradient);
    //we need to scale this.
    scale_p<<<1,NT>>>(ce.d_comp_kernel_grad,ce.filter_size);
    hipDeviceSynchronize();
    to_real_p<<<1,NT>>>(ce.d_comp_kernel_grad,ce.d_kernel_grad,ce.filter_size);
    hipDeviceSynchronize();
    updatekernel<<<1,NT>>>(ce.d_kernel,ce.d_kernel_grad,ce.learning_rate,ce.filter_size);
    hipDeviceSynchronize();
}
void fixKernel(convolution_enviorment ce){
    hipMemcpy(ce.d_kernel,ce.kernel,ce.filter_size*ce.filter_size,hipMemcpyHostToDevice);
}
void forwardPass_parallel(convolution_enviorment ce){
    //cout<<"starting the routine"<<endl;
    int fs=ce.filter_size*ce.filter_size;
    hipMemcpy(ce.d_layer,ce.layer,fs,hipMemcpyHostToDevice);
    //cudaMemcpy(ce.d_kernel,ce.kernel,fs,cudaMemcpyHostToDevice);
    //cout<<"senf the kernel and layer to the device"<<endl;
    conv_complex<<<1,NT>>>(ce.d_layer,ce.d_comp_layer,ce.filter_size);
    conv_complex<<<1,NT>>>(ce.d_kernel,ce.d_comp_kernel,ce.filter_size);
    //cout<<"convertin them to complex"<<endl;
    ce.get_layer=fftwf_plan_dft_2d(ce.filter_size,ce.filter_size,ce.d_comp_layer,ce.d_fft_layer,FFTW_FORWARD,FFTW_ESTIMATE);
    ce.get_kernel=fftwf_plan_dft_2d(ce.filter_size,ce.filter_size,ce.d_comp_kernel,ce.d_fft_kernel,FFTW_FORWARD,FFTW_ESTIMATE);
    fftwf_execute(ce.get_layer);
    fftwf_execute(ce.get_kernel);    
    hipDeviceSynchronize();
    //cout<<"doing the fft transform"<<endl;
    poin_mul_parallel<<<1,NT>>>(ce.d_fft_kernel,ce.d_fft_layer,ce.d_fft_mul,ce.filter_size);
    hipDeviceSynchronize();
    //cout<<"done point wise multiplication"<<endl;
    ce.getBack_mul=fftwf_plan_dft_2d(ce.filter_size,ce.filter_size,ce.d_fft_mul,ce.d_mul,FFTW_BACKWARD,FFTW_ESTIMATE);
    fftwf_execute(ce.getBack_mul);
    hipDeviceSynchronize();
    //cout<<"inverse fft done"<<endl;
    scale_p<<<1,NT>>>(ce.d_mul,ce.filter_size);   
    hipDeviceSynchronize();
    //cout<<"scalling is done"<<endl;
    to_real_p<<<1,NT>>>(ce.d_mul,ce.d_real_mul,ce.filter_size);
    hipDeviceSynchronize();
    //cout<<"convertion to real done"<<endl;
    sumP<<<1,NT>>>(ce.d_real_mul,ce.d_predicted,ce.filter_size);
    hipMemcpy(ce.predicted,ce.d_predicted,sizeof(double),hipMemcpyDeviceToHost);
    //cout<<"summation doen and copy to host done"<<endl;
}
void trainIteration(convolution_enviorment ce){
    forwardPass_parallel(ce);
    backPropogate(ce,30);
}

void mat_mul(double* a,double* b,double* res,int n){
    for (int i = 0; i<n;i++) {
        for (int j =0; j<n;j++) {
            for (int k=0;k<n;k++) {
                res[ind_ele(n,i,j)]+=a[ind_ele(n,i,k)]*b[ind_ele(n,k,j)];
            }
        }
    }
}




void messaure_normal_time(){
    int layer_length=512;
    double* layer=create_matrix(layer_length);
    fill_random(layer,layer_length);
    double* output=create_matrix(layer_length);
    for(int kernel_length=3;kernel_length<512;kernel_length+=2){
        double* kernel=create_matrix(kernel_length);
        fill_random(kernel,kernel_length);
        clock_t start_time;
        clock_t end_time;
        start_time=clock();
        for(int i=0;i<10;i++){
            conv(layer,kernel,layer_length,kernel_length,output);
        }
        end_time=clock();
        double clock_taken=double(end_time - start_time);
        clock_taken=clock_taken/10;
        double time_taken=clock_taken/double(CLOCKS_PER_SEC);
        cout<<kernel_length<<" "<<time_taken<<endl;
        delete kernel;
    }
}

//before we need to set the kernel we need to preprocess teh kernel
//we need to set the layer
//data set is not coded in it yet



int main(){
    cout<<"enter layer size"<<endl;
    int l_s;
    cin>>l_s;
    int k_s;
    cout<<"enter kernel size"<<endl;
    cin>>k_s;
    if(k_s%2==0){
        cout<<"kernel should be odd"<<endl;
        return 0;
    }
    convolution_enviorment ce=create_convolution_enviormentParellel(l_s);
    ini_bp(&ce,0.00000000001);
    double* k=create_matrix(k_s);
    fill_random(ce.layer,l_s);
    fill_constant(k,20,k_s);
    preProcessKernel_static(k,ce.kernel,k_s,l_s);
    fixKernel(ce);

    clock_t start_time;
    clock_t end_time;
    start_time=clock();
    for(int i=0;i<120;i++)
    trainIteration(ce);
    end_time=clock();
    double clock_taken=double(end_time - start_time);
    double time_taken=clock_taken/double(CLOCKS_PER_SEC);
    cout<<"the time taken is "<<time_taken<<endl;
    return 0;
}
//layer=[[1,1,1,2,1],[8,3,2,6,2],[4,5,6,3,3],[1,1,1,1,1],[3,3,3,4,5]]
//kernel=[[1,1,2],[3,1,2],[4,5,5]]
