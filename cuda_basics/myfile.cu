
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
__global__ void  mykernel(int* a,int* b,int* c){
    //no code is here
    *c=*a+*b;
}
int main(){
    int a=1;
    int b=9;
    int c;
    int* d_a;
    int* d_b;
    int* d_c;
    hipMalloc((void**)&d_a,sizeof(int));
    hipMalloc((void**)&d_b,sizeof(int));
    hipMalloc((void**)&d_c,sizeof(int));
    hipMemcpy(d_a,&a,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,&b,sizeof(int),hipMemcpyHostToDevice);
    mykernel<<<1,1>>>(d_a,d_b,d_c);
    hipMemcpy(&c,d_c,sizeof(int),hipMemcpyDeviceToHost);
    printf("the summation is %d",c);
    return 0;
}
/*
#include<stdio.h>
__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<1,1>>>(); 
    printf("hello world from host");
    return 0;
}*/